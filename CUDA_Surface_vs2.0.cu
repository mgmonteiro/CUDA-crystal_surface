#include<math.h>
#include<stdlib.h>
#include<stdio.h>
#include<string.h>
#include<hip/hip_runtime.h>
#include<time.h>


//CUDA error wrapping

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

/*
**************************************************************************************************
Author: Maxwel Gama Monteiro Junior
Contact: maxweljr@gmail.com

Description: Obtains the surface of a given crystal using its coordination number as order
parameter.
**************************************************************************************************
                       ;\
                      _' \_
                    ,' '  '`.
                   ;,)       \
                  /          :
                  (_         :
                   `--.       \			~ CUDA Crystal Surface Cutter vs2.0
                      /        `.

Version History:

1.0 - Does stuff

2.0 - Does stuff WITH MORE GLITTER! oh and also carries on counting on GPU only - see
deprecated function at bottom

**************************************************************************************************
**************************************************************************************************
**************************************************************************************************
*/

//Double approximated reciprocal square root function (drsqrt)

    __device__ __forceinline__ double drsqrt (double a)
    {
      double y, h, l, e;
      unsigned int ilo, ihi, g, f;
      int d;

      ihi = __double2hiint(a);
      ilo = __double2loint(a);
      if (((unsigned int)ihi) - 0x00100000U < 0x7fe00000U){
        f = ihi | 0x3fe00000;
        g = f & 0x3fffffff;
        d = g - ihi;
        a = __hiloint2double(g, ilo); 
        y = rsqrt (a);
        h = __dmul_rn (y, y);
        l = __fma_rn (y, y, -h);
        e = __fma_rn (l, -a, __fma_rn (h, -a, 1.0));
        // Round as shown in Peter Markstein, "IA-64 and Elementary Functions"
        y = __fma_rn (__fma_rn (0.375, e, 0.5), e * y, y);
        d = d >> 1;
        a = __hiloint2double(__double2hiint(y) + d, __double2loint(y));
      } else if (a == 0.0) {
        a = __hiloint2double ((ihi & 0x80000000) | 0x7ff00000, 0x00000000);
      } else if (a < 0.0) {
        a = __hiloint2double (0xfff80000, 0x00000000);
      } else if (isinf (a)) {
        a = __hiloint2double (ihi & 0x80000000, 0x00000000);
      } else if (isnan (a)) {
        a = a + a;
      } else {
        a = a * __hiloint2double (0x7fd00000, 0);
        y = rsqrt (a);
        h = __dmul_rn (y, y);
        l = __fma_rn (y, y, -h);
        e = __fma_rn (l, -a, __fma_rn (h, -a, 1.0));
        // Round as shown in Peter Markstein, "IA-64 and Elementary Functions"
        y = __fma_rn (__fma_rn (0.375, e, 0.5), e * y, y);
        a = __hiloint2double(__double2hiint(y) + 0x1ff00000,__double2loint(y));
      }
      return a;
    }


__global__ void parameter_counter(double *x_, double *y_, double *z_, int *count, uint atoms, double prmt_, uint *surfies_)
{
	int n = threadIdx.x + blockDim.x * blockIdx.x;

	extern __shared__ uint cache[];	
	uint temp = 0;
	

while (n < atoms)
{
	double xx_;
	double yy_;
	double zz_;
	double rij_;
	double x = x_[n];
	double y = y_[n];
	double z = z_[n];
	int kappa = 0; //save to thread register instead of reordering *count
	

		for (int neighbor = 0; neighbor < atoms; neighbor++)
		{
				xx_ =  x - x_[neighbor];
				yy_ = (y - y_[neighbor])*(y - y_[neighbor]);
				zz_ = (z - z_[neighbor])*(z - z_[neighbor]);
				rij_ = drsqrt(xx_*xx_ + yy_ + zz_);

			  if (1.0/rij_ < prmt_) 
			  {
				kappa++; //Count the atom itself to avoid branching 
			  }
			//if (1.0/rij_ < prmt_ && isinf(rij_) == 0)kappa++;//Check that one is not his own neighbor
		}

count[n] = kappa;

if(kappa < 13){
	temp++;
}  //Surface atoms will have less than the usual 12 neighbors of FCC (13 in this case because it is self-interacting)


n +=gridDim.x * blockDim.x;
}


cache[threadIdx.x] = temp;

__syncthreads();

//Perform sum reduction on temp values to obtain number of surface atoms

int u = blockDim.x/2;

while(u != 0)
	{
		if (threadIdx.x < u)
			{
			cache[threadIdx.x] += cache[threadIdx.x + u];
			}
	__syncthreads();
	u /= 2;
	}

if (threadIdx.x == 0) surfies_[blockIdx.x] = cache[0];



}



int main(void)
{

	int deviceCount;
	hipGetDeviceCount (&deviceCount);
	if (deviceCount <1)
	{
		printf("CUDA supporting video card not detected. Go eat a sandwich or something.");
		return 0;
	}
	
	

	double *x, *y, *z, *dev_x, *dev_y, *dev_z;
	uint *surfies, *dev_surfies;
	int *count, *dev_count;

	int *lbl;
	double prmt;
	int natom;
	int i, j, p;
	int n_surface = 0;

	//Adjust the size of histogram accordingly to fit your maximum number of nearest neighbors
	int grp[14];

	//Change as suitable, this seems to work reasonably in general
	size_t block = 512;
	size_t thread = 512;

	size_t block_size = sizeof(uint)*block;

	


	FILE *finp, *fout, *fsupply;
	finp=fopen("coord_z.xyz","r");
	fsupply=fopen("num_edge.dat","w"); 
	fout=fopen("surface.xyz","w");
	
	fscanf(finp,"%d\n",&natom);
	fscanf(finp,"%lf\n",&prmt);
	

	hipHostMalloc((void**)&lbl, sizeof(int) * natom, hipHostMallocDefault);
	hipHostMalloc((void**)&x,sizeof(double) * natom, hipHostMallocDefault);
	hipHostMalloc((void**)&y,sizeof(double) * natom, hipHostMallocDefault);
	hipHostMalloc((void**)&z,sizeof(double) * natom, hipHostMallocDefault);
	hipHostMalloc((void**)&count,sizeof(int) * natom, hipHostMallocDefault);
	hipHostMalloc((void**)&surfies, block_size, hipHostMallocDefault);

	gpuErrchk( hipMalloc((void**)&dev_surfies, block_size)	      );
	gpuErrchk( hipMalloc((void**)&dev_x, sizeof(double) * natom) );
	gpuErrchk( hipMalloc((void**)&dev_y, sizeof(double) * natom) );
	gpuErrchk( hipMalloc((void**)&dev_z, sizeof(double) * natom) );
	gpuErrchk( hipMalloc((void**)&dev_count, sizeof(int) * natom));

	j = 0;
	while (fscanf(finp,"%d %lf %lf %lf\n",&lbl[j], &x[j], &y[j], &z[j]) == 4)
	{

	j=j+1;

	}

	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start,0);

	hipMemcpyAsync(dev_x, x, sizeof(double)*natom, hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_y, y, sizeof(double)*natom, hipMemcpyHostToDevice);
	hipMemcpyAsync(dev_z, z, sizeof(double)*natom, hipMemcpyHostToDevice);
	
	for (p = 0; p < 14; p++)
	{
		grp[p] = 0;
	}

	prmt *= 0.8; //Lattice parameter/sqrt(2) ~ prmt * 0.7 is the nearest neighbor FCC distance, using 0.8 to make sure every neighbor is contained
		     //even if fast-math flags are used

	parameter_counter<<<thread, block, block_size>>>(dev_x,dev_y,dev_z,dev_count,natom, prmt, dev_surfies);
	
	hipMemcpyAsync(count, dev_count, sizeof(int)*natom, hipMemcpyDeviceToHost);
	hipMemcpy(surfies, dev_surfies, block_size, hipMemcpyDeviceToHost); //This is needed immediately by host


	for(p = 0; p < block; p++){
	n_surface+=surfies[p];
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, start, stop); 

	clock_t launch, finish;

	double cpu_time;
	launch = clock();


		fprintf(fout,"%d\n",n_surface);
		fprintf(fout,"%16.16lf\n",prmt/0.8);	

	for (i = 0; i < natom; i++) //Writing and wrap-up done on host
		{
			j = count[i];
			grp[j]++;
			
			if(count[i] < 13){

			fprintf(fout,"%d \t %16.15lf \t %16.15lf \t %16.15lf \n",lbl[i], x[i], y[i], z[i]);
			}
		}

	
	j = 0;
	for (i = 0; i < 14; i++)
		{
			fprintf(fsupply, "%d\n", grp[i] );
			j+=grp[i];
		}
		
			fprintf(fsupply,"%d\n",j);



	hipHostFree(x);
	hipHostFree(y);
	hipHostFree(z);
	hipHostFree(count);
	hipHostFree(surfies);
	hipHostFree(lbl);
	hipFree(dev_x);
	hipFree(dev_y);
	hipFree(dev_z);
	hipFree(dev_count);
	hipFree(dev_surfies);

finish = clock();
cpu_time = ((double)(finish - launch)) / CLOCKS_PER_SEC;

fclose(finp);
fclose(fout);
fclose(fsupply);


printf("\n\nCPU process finished at %16.8lf seconds\n\n", cpu_time);
printf("\n\nGPU process finished at %.8f seconds\n\n", elapsedTime/1000);
printf("======================================================================~\n");


return 0;
}

